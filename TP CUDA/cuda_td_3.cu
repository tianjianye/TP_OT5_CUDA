#include "hip/hip_runtime.h"
#include "wb.h"

#define wbCheck(stmt)\
    do {\
    hipError_t err = stmt;\
    if (err != hipSuccess) {\
        wbLog(ERROR, "Failed to run stmt ", #stmt);\
        wbLog(ERROR, "Got CUDA error ... ", hipGetErrorString(err));\
        return -1;\
    }\
} while (0)\

#define BLUR_SIZE 5

//@@ INSERT CODE HERE

__global__ void blurKernel(int height, int width,float *input, float *output) {    
	int Row=threadIdx.x+blockIdx.x*blockDim.x;	
	int Col=threadIdx.y+blockIdx.y*blockDim.y;
	if(Col<width && Row<height){
		for(int k=0;k<3;++k){
			float pixVal=0;
			int pixels=0;
			for(int blurRow=-BLUR_SIZE;blurRow<BLUR_SIZE+1;++blurRow){	
				for(int blurCol=-BLUR_SIZE;blurCol<BLUR_SIZE+1;++blurCol){	
					int curRow=Row+blurRow;
					int curCol=Col+blurCol;
					if(curRow>-1 && curRow<height && curCol>-1 && curCol<width){
						pixVal+=input[(curRow*width+curCol)*3+k];
						pixels++;			
					}
				}
			}	
			output[(Row*width+Col)*3+k]=pixVal/pixels;	
		}
	}
}

int main(int argc, char *argv[]) {
    wbArg_t args;
    int imageWidth;
    int imageHeight;
    char *inputImageFile;
	char *outputImageFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float *hostInputImageData;
    float *hostOutputImageData;
    float *deviceInputImageData;
    float *deviceOutputImageData;
    args = wbArg_read(argc, argv); /* parse the input arguments */
    inputImageFile = wbArg_getInputFile(args, 0);
	outputImageFile = wbArg_getInputFile(args, 1);
    inputImage = wbImport(inputImageFile);
    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    outputImage = wbImage_new(imageWidth, imageHeight, 3);
    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);
    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **)&deviceInputImageData,
    imageWidth * imageHeight * sizeof(float)*3);
    hipMalloc((void **)&deviceOutputImageData,
    imageWidth * imageHeight * sizeof(float)*3);
    wbTime_stop(GPU, "Doing GPU memory allocation");
    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputImageData, hostInputImageData,
    imageWidth * imageHeight * sizeof(float) * 3,
    hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");
    ///////////////////////////////////////////////////////
    wbTime_start(Compute, "Doing the computation on the GPU");

	dim3 DimGrid((imageHeight-1.0)/16.0+1.0,(imageWidth-1.0)/16.0+1.0,1);
	dim3 DimBlock(16,16,1);
	blurKernel<<<DimGrid, DimBlock>>>(imageHeight, imageWidth, deviceInputImageData, deviceOutputImageData);	
	
    wbTime_stop(Compute, "Doing the computation on the GPU");
    ///////////////////////////////////////////////////////
    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutputImageData, deviceOutputImageData,
    imageWidth * imageHeight * sizeof(float) * 3,
    hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");
    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");
    
    int i, j;
    FILE *fp = fopen(outputImageFile, "wb"); /* b - binary mode */
    (void) fprintf(fp, "P6\n%d %d\n255\n", imageWidth, imageHeight);
        for (i = 0; i < imageHeight; ++i)
    {
    for (j = 0; j < imageWidth; ++j)
        {
            static unsigned char color[3];
            color[0] = hostOutputImageData[(i*imageWidth+j)*3]*255;  /* red */
            color[1] = hostOutputImageData[(i*imageWidth+j)*3+1]*255;  /* green */
            color[2] = hostOutputImageData[(i*imageWidth+j)*3+2]*255;  /* blue */

            (void) fwrite(color, 1, 3, fp);
        }
    }
    (void) fclose(fp);
    
    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);
    return 0;
}
